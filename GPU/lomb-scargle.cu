#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

int const blocksize = 512;


__global__
void periodogram_frequency( double *time, double *flux, double *frequency, double *periodogram, double one_over_2variance, size_t sample_size, size_t num_frequencies){
    // Calculate frequency index

    //One thing, printf works from the kernel but only up to a certain length then it just doesn't print anymore. Learnt that the hard way...
    int const w = threadIdx.x + blockIdx.x * blockDim.x;
    if ( w < num_frequencies) { 
       
        double cos_sum_squared = 0llu, cos_squared_sum = 0llu, sin_sum_squared = 0llu, sin_squared_sum = 0llu;
       
        
        double frequency_2 = 2 * frequency[w];
   
        //A potential way forward might be to split the frequencies across blocks and this across threads? I had to remove the temp arrays as there wasn't enough memory
        //Also not really sure we need them anymore as they were mainly for vectorization
        for(int i = 0; i< sample_size;i++){
            float coscal;
            float sincal;
            sincospi(frequency_2 * time[i],&sincal,&coscal);
            cos_sum_squared = fma(flux[i],(double) coscal,cos_sum_squared);
            cos_squared_sum = fma((double) coscal,(double) coscal,cos_squared_sum);
            sin_sum_squared = fma(flux[i],(double) sincal,sin_sum_squared);
            sin_squared_sum = fma((double) sincal,(double) sincal,sin_squared_sum);
        }
        cos_sum_squared = cos_sum_squared * cos_sum_squared;
        sin_sum_squared = sin_sum_squared * sin_sum_squared;

        periodogram[w] = one_over_2variance * (cos_sum_squared/cos_squared_sum + sin_sum_squared/sin_squared_sum);
    }
}



void lomb_scargle(double *flux, double *time, double *frequency, double *periodogram, double variance, size_t sample_size, size_t num_frequencies){

    double *dev_frequency, *dev_periodogram;
    thrust::device_vector<double> time2(time,time+sample_size);
    thrust::device_vector<double> flux2(flux,flux+sample_size);
 
    hipMalloc( (void **) &dev_frequency, num_frequencies * sizeof(double));
    hipMalloc( (void **) &dev_periodogram, num_frequencies * sizeof(double));

    hipMemcpy( dev_frequency, frequency, num_frequencies * sizeof(double), hipMemcpyHostToDevice );

    
    double flux_avg = thrust::reduce(flux2.begin(), flux2.end())/sample_size;
    using namespace thrust::placeholders;
    thrust::transform(flux2.begin(), flux2.end(), flux2.begin(), _1 - flux_avg); 
    thrust::transform(time2.begin(), time2.end(), time2.begin(), _1 - time[0]); 

    double one_over_2variance = 1 / (2 * variance);
    auto const num_blocks = ceil(num_frequencies/ static_cast< float >(blocksize));
 
    periodogram_frequency<<< num_blocks, blocksize >>>(thrust::raw_pointer_cast(time2.data()), thrust::raw_pointer_cast(flux2.data()), dev_frequency, dev_periodogram, one_over_2variance, sample_size, num_frequencies);
    
    //Cuda fails silently, need this to see errors
    hipError_t err2 = hipGetLastError();
    if (err2 != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err2));
    hipMemcpy( periodogram, dev_periodogram, num_frequencies * sizeof(double), hipMemcpyDeviceToHost );

    hipFree(dev_frequency);
    hipFree(dev_periodogram);
}

